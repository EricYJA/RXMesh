#include "hip/hip_runtime.h"
// Parallel version of
// Desbrun, Mathieu, et al "Implicit Fairing of Irregular Meshes using Diffusion
// and Curvature Flow." SIGGRAPH 1999

#include <omp.h>

#include "gtest/gtest.h"
#include "rxmesh/attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/cuda_query.h"
#include "rxmesh/util/export_tools.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/log.h"

struct arg
{
    std::string obj_file_name       = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder       = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id           = 0;
    float       time_step           = 0.001;
    float       cg_tolerance        = 1e-6;
    uint32_t    max_num_cg_iter     = 1000;
    bool        use_uniform_laplace = false;
    uint32_t    nd_level            = 4;
    char**      argv;
    int         argc;
} Arg;

#include "mcf_cg.h"
#include "mcf_cusolver_chol.cuh"


TEST(App, MCF)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);

    RXMeshStatic rx(Arg.obj_file_name);

    // RXMesh Impl
    mcf_cg<dataT>(rx);

    // RXMesh cusolver Impl with our CUDA_ND reorder
    mcf_cusolver_chol_cudaND<dataT>(rx);

    // RXMesh cusolver Impl
    mcf_cusolver_chol<dataT>(rx);
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;
    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: MCF.exe < -option X>\n"
                        " -h:                 Display this massage and exit\n"
                        " -input:             Input file. Input file should be under the input/ subdirectory\n"
                        "                     Default is {} \n"
                        "                     Hint: Only accept OBJ files\n"
                        " -o:                 JSON file output folder. Default is {} \n"
                        " -uniform_laplace:   Use uniform Laplace weights. Default is {} \n"
                        " -dt:                Time step (delta t). Default is {} \n"
                        "                     Hint: should be between (0.001, 1) for cotan Laplace or between (1, 100) for uniform Laplace\n"
                        " -eps:               Conjugate gradient tolerance. Default is {}\n"
                        " -max_cg_iter:       Conjugate gradient maximum number of iterations. Default is {}\n"
                        " -nd_level:          ND level. Default is {}\n"                        
                        " -device_id:         GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder,  (Arg.use_uniform_laplace? "true" : "false"), Arg.time_step, Arg.cg_tolerance, Arg.max_num_cg_iter, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-dt")) {
            Arg.time_step = std::atof(get_cmd_option(argv, argv + argc, "-dt"));
        }
        if (cmd_option_exists(argv, argc + argv, "-max_cg_iter")) {
            Arg.max_num_cg_iter =
                std::atoi(get_cmd_option(argv, argv + argc, "-max_cg_iter"));
        }

        if (cmd_option_exists(argv, argc + argv, "-eps")) {
            Arg.cg_tolerance =
                std::atof(get_cmd_option(argv, argv + argc, "-eps"));
        }
        if (cmd_option_exists(argv, argc + argv, "-uniform_laplace")) {
            Arg.use_uniform_laplace = true;
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
        if (cmd_option_exists(argv, argc + argv, "-nd_level")) {
            Arg.nd_level =
                atoi(get_cmd_option(argv, argv + argc, "-nd_level"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("max_num_cg_iter= {}", Arg.max_num_cg_iter);
    RXMESH_TRACE("cg_tolerance= {0:f}", Arg.cg_tolerance);
    RXMESH_TRACE("use_uniform_laplace= {}", Arg.use_uniform_laplace);
    RXMESH_TRACE("time_step= {0:f}", Arg.time_step);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}