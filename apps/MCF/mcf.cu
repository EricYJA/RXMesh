#include "hip/hip_runtime.h"
// Parallel version of
// Desbrun, Mathieu, et al "Implicit Fairing of Irregular Meshes using Diffusion
// and Curvature Flow." SIGGRAPH 1999

#include <omp.h>

#include "../common/openmesh_trimesh.h"
#include "gtest/gtest.h"
#include "rxmesh/attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/cuda_query.h"
#include "rxmesh/util/export_tools.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/log.h"

struct arg
{
    std::string obj_file_name       = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder       = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id           = 0;
    float       time_step           = 0.001;
    float       cg_tolerance        = 1e-6;
    uint32_t    max_num_cg_iter     = 1000;
    bool        use_uniform_laplace = false;
    char**      argv;
    int         argc;
} Arg;

#include "mcf_openmesh.h"
#include "mcf_rxmesh.h"
#include "mcf_sparse_matrix.cuh"


TEST(App, MCF)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);

    RXMeshStatic rxmesh(Arg.obj_file_name, false);

    TriMesh input_mesh;
    ASSERT_TRUE(OpenMesh::IO::read_mesh(input_mesh, Arg.obj_file_name));


    // OpenMesh Impl
    std::vector<std::vector<dataT>> ground_truth(rxmesh.get_num_vertices());
    for (auto& g : ground_truth) {
        g.resize(3);
    }
    // mcf_openmesh(omp_get_max_threads(), input_mesh, ground_truth);

    // // RXMesh Impl
    // mcf_rxmesh_cg(rxmesh, ground_truth);  

    // RXMesh cusolver Impl
    mcf_rxmesh_cusolver_chol(rxmesh, ground_truth, Arg.obj_file_name); 
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;
    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: MCF.exe < -option X>\n"
                        " -h:                 Display this massage and exit\n"
                        " -input:             Input file. Input file should be under the input/ subdirectory\n"
                        "                     Default is {} \n"
                        "                     Hint: Only accept OBJ files\n"
                        " -o:                 JSON file output folder. Default is {} \n"
                        " -uniform_laplace:   Use uniform Laplace weights. Default is {} \n"
                        " -dt:                Time step (delta t). Default is {} \n"
                        "                     Hint: should be between (0.001, 1) for cotan Laplace or between (1, 100) for uniform Laplace\n"
                        " -eps:               Conjugate gradient tolerance. Default is {}\n"
                        " -max_cg_iter:       Conjugate gradient maximum number of iterations. Default is {}\n"                        
                        " -device_id:         GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder,  (Arg.use_uniform_laplace? "true" : "false"), Arg.time_step, Arg.cg_tolerance, Arg.max_num_cg_iter, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-dt")) {
            Arg.time_step = std::atof(get_cmd_option(argv, argv + argc, "-dt"));
        }
        if (cmd_option_exists(argv, argc + argv, "-max_cg_iter")) {
            Arg.max_num_cg_iter =
                std::atoi(get_cmd_option(argv, argv + argc, "-max_cg_iter"));
        }

        if (cmd_option_exists(argv, argc + argv, "-eps")) {
            Arg.cg_tolerance =
                std::atof(get_cmd_option(argv, argv + argc, "-eps"));
        }
        if (cmd_option_exists(argv, argc + argv, "-uniform_laplace")) {
            Arg.use_uniform_laplace = true;
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("max_num_cg_iter= {}", Arg.max_num_cg_iter);
    RXMESH_TRACE("cg_tolerance= {0:f}", Arg.cg_tolerance);
    RXMESH_TRACE("use_uniform_laplace= {}", Arg.use_uniform_laplace);
    RXMESH_TRACE("time_step= {0:f}", Arg.time_step);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}