#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include "rxmesh/attribute.h"
#include "rxmesh/matrix/dense_matrix.cuh"
#include "rxmesh/matrix/sparse_matrix.cuh"
#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_static.h"

#include <Eigen/SparseCholesky>

template <uint32_t blockThreads, typename IndexT = int>
__global__ static void sparse_mat_test(const rxmesh::Context context,
                                       IndexT*               vet_degree)
{
    using namespace rxmesh;
    auto compute_valence = [&](VertexHandle& v_id, const VertexIterator& iter) {
        auto     ids      = v_id.unpack();
        uint32_t patch_id = ids.first;
        uint16_t local_id = ids.second;
        vet_degree[context.vertex_prefix()[patch_id] + local_id] =
            iter.size() + 1;
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, compute_valence);
}

template <typename T, uint32_t blockThreads>
__global__ static void sparse_mat_edge_len_test(
    const rxmesh::Context      context,
    rxmesh::VertexAttribute<T> coords,
    rxmesh::SparseMatrix<T>    sparse_mat,
    T*                         arr_ref)
{
    using namespace rxmesh;
    auto compute_edge_len = [&](VertexHandle&         v_id,
                                const VertexIterator& iter) {
        // reference value calculation
        auto     r_ids      = v_id.unpack();
        uint32_t r_patch_id = r_ids.first;
        uint16_t r_local_id = r_ids.second;

        uint32_t row_index = context.vertex_prefix()[r_patch_id] + r_local_id;

        arr_ref[row_index]     = 0;
        sparse_mat(v_id, v_id) = 0;

        vec3<T> v_coord(coords(v_id, 0), coords(v_id, 1), coords(v_id, 2));
        for (uint32_t v = 0; v < iter.size(); ++v) {
            vec3<T> vi_coord(
                coords(iter[v], 0), coords(iter[v], 1), coords(iter[v], 2));

            sparse_mat(v_id, iter[v]) = 1;  // dist(v_coord, vi_coord);

            arr_ref[row_index] += 1;  // dist(v_coord, vi_coord);
        }
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, compute_edge_len);
}

template <typename T>
__global__ void spmat_multi_hardwired_kernel(T*                      vec,
                                             rxmesh::SparseMatrix<T> sparse_mat,
                                             T*                      out,
                                             const int               N)
{
    int   tid = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0;
    if (tid < N) {
        uint32_t start = sparse_mat.row_ptr()[tid];
        uint32_t end   = sparse_mat.row_ptr()[tid + 1];
        for (int i = 0; i < end - start; i++) {
            sum += vec[sparse_mat.col_idx()[start + i]] *
                   sparse_mat.get_val_at(start + i);
        }
        out[tid] = sum;
    }
}

template <typename T, uint32_t blockThreads>
__global__ static void simple_A_X_B_setup(const rxmesh::Context      context,
                                          rxmesh::VertexAttribute<T> coords,
                                          rxmesh::SparseMatrix<T>    A_mat,
                                          rxmesh::DenseMatrix<T>     X_mat,
                                          rxmesh::DenseMatrix<T>     B_mat,
                                          const T                    time_step)
{
    using namespace rxmesh;
    auto mat_setup = [&](VertexHandle& v_id, const VertexIterator& iter) {
        T sum_e_weight(0);

        T v_weight = iter.size();

        // reference value calculation

        B_mat(v_id, 0) = iter.size() * 7.4f;
        B_mat(v_id, 1) = iter.size() * 2.6f;
        B_mat(v_id, 2) = iter.size() * 10.3f;

        X_mat(v_id, 0) = coords(v_id, 0) * v_weight;
        X_mat(v_id, 1) = coords(v_id, 1) * v_weight;
        X_mat(v_id, 2) = coords(v_id, 2) * v_weight;

        vec3<T> vi_coord(coords(v_id, 0), coords(v_id, 1), coords(v_id, 2));
        for (uint32_t v = 0; v < iter.size(); ++v) {
            T e_weight           = 1;
            A_mat(v_id, iter[v]) = time_step * e_weight;

            sum_e_weight += e_weight;
        }

        A_mat(v_id, v_id) = v_weight + time_step * sum_e_weight +
                            iter.size() * iter.size() + 1000000;
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, mat_setup);
}


TEST(RXMeshStatic, SparseMatrix)
{
    // Test accessing of the sparse matrix in CSR format in device

    using namespace rxmesh;

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    uint32_t num_vertices = rx.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    int* d_arr_ones;
    int* d_result;

    std::vector<int> init_tmp_arr(num_vertices, 1);
    CUDA_ERROR(hipMalloc((void**)&d_arr_ones, (num_vertices) * sizeof(int)));
    CUDA_ERROR(hipMemcpy(d_arr_ones,
                          init_tmp_arr.data(),
                          num_vertices * sizeof(int),
                          hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&d_result, (num_vertices) * sizeof(int)));

    SparseMatrix<int> spmat(rx);
    spmat.reset(1, LOCATION_ALL);

    spmat_multi_hardwired_kernel<<<blocks, threads>>>(
        d_arr_ones, spmat, d_result, num_vertices);

    std::vector<int> h_result(num_vertices);
    CUDA_ERROR(hipMemcpy(
        h_result.data(), d_result, num_vertices, hipMemcpyDeviceToHost));

    // get reference result
    int* vet_degree;
    CUDA_ERROR(hipMalloc((void**)&vet_degree, (num_vertices) * sizeof(int)));

    LaunchBox<threads> launch_box;
    rx.prepare_launch_box(
        {Op::VV}, launch_box, (void*)sparse_mat_test<threads>);

    // test kernel
    sparse_mat_test<threads>
        <<<launch_box.blocks,
           launch_box.num_threads,
           launch_box.smem_bytes_dyn>>>(rx.get_context(), vet_degree);

    std::vector<int> h_vet_degree(num_vertices);
    CUDA_ERROR(hipMemcpy(
        h_vet_degree.data(), vet_degree, num_vertices, hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        EXPECT_EQ(h_result[i], h_vet_degree[i]);
    }


    CUDA_ERROR(hipFree(d_arr_ones));
    CUDA_ERROR(hipFree(d_result));
    CUDA_ERROR(hipFree(vet_degree));
    spmat.release();
}

TEST(RXMeshStatic, SparseMatrixEdgeLen)
{
    // First replace the sparse matrix entry with the edge length and then do
    // spmv with an all one array and check the result
    //
    using namespace rxmesh;


    // generate rxmesh obj
    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");

    uint32_t num_vertices = rx.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto coords = rx.get_input_vertex_coordinates();

    float* d_arr_ones;

    std::vector<float> init_tmp_arr(num_vertices, 1.f);
    CUDA_ERROR(hipMalloc((void**)&d_arr_ones, (num_vertices) * sizeof(float)));
    CUDA_ERROR(hipMemcpy(d_arr_ones,
                          init_tmp_arr.data(),
                          num_vertices * sizeof(float),
                          hipMemcpyHostToDevice));

    SparseMatrix<float> spmat(rx);

    float* d_arr_ref;
    float* d_result;

    CUDA_ERROR(hipMalloc((void**)&d_arr_ref, (num_vertices) * sizeof(float)));
    CUDA_ERROR(hipMalloc((void**)&d_result, (num_vertices) * sizeof(float)));

    LaunchBox<threads> launch_box;
    rx.prepare_launch_box(
        {Op::VV}, launch_box, (void*)sparse_mat_edge_len_test<float, threads>);

    sparse_mat_edge_len_test<float, threads><<<launch_box.blocks,
                                               launch_box.num_threads,
                                               launch_box.smem_bytes_dyn>>>(
        rx.get_context(), *coords, spmat, d_arr_ref);

    spmat.multiply(d_arr_ones, d_result);

    // copy the value back to host
    std::vector<float> h_arr_ref(num_vertices);
    CUDA_ERROR(hipMemcpy(h_arr_ref.data(),
                          d_arr_ref,
                          num_vertices * sizeof(float),
                          hipMemcpyDeviceToHost));

    std::vector<float> h_result(num_vertices);
    CUDA_ERROR(hipMemcpy(h_result.data(),
                          d_result,
                          num_vertices * sizeof(float),
                          hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        EXPECT_FLOAT_EQ(h_result[i], h_arr_ref[i]);
    }

    CUDA_ERROR(hipFree(d_arr_ref));
    CUDA_ERROR(hipFree(d_arr_ones));
    CUDA_ERROR(hipFree(d_result));
    spmat.release();
}

TEST(RXMeshStatic, SparseMatrixSimpleSolve)
{
    // set up a simple AX=B system where A is a sparse matrix, B and C are dense
    // matrix.

    using namespace rxmesh;

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");

    uint32_t num_vertices = rx.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto                coords = rx.get_input_vertex_coordinates();
    SparseMatrix<float> A_mat(rx);
    DenseMatrix<float>  X_mat(rx, num_vertices, 3);
    DenseMatrix<float>  B_mat(rx, num_vertices, 3);
    DenseMatrix<float>  ret_mat(rx, num_vertices, 3);

    float time_step = 1.f;

    LaunchBox<threads> launch_box;
    rx.prepare_launch_box(
        {Op::VV}, launch_box, (void*)simple_A_X_B_setup<float, threads>);

    simple_A_X_B_setup<float, threads><<<launch_box.blocks,
                                         launch_box.num_threads,
                                         launch_box.smem_bytes_dyn>>>(
        rx.get_context(), *coords, A_mat, X_mat, B_mat, time_step);

    A_mat.solve(B_mat, X_mat, Solver::CHOL, PermuteMethod::NSTDIS);


    A_mat.multiply(X_mat, ret_mat);


    std::vector<vec3<float>> h_ret_mat(num_vertices);
    CUDA_ERROR(hipMemcpy(h_ret_mat.data(),
                          ret_mat.data(),
                          num_vertices * 3 * sizeof(float),
                          hipMemcpyDeviceToHost));
    std::vector<vec3<float>> h_B_mat(num_vertices);
    CUDA_ERROR(hipMemcpy(h_B_mat.data(),
                          B_mat.data(),
                          num_vertices * 3 * sizeof(float),
                          hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        for (uint32_t j = 0; j < 3; ++j) {
            EXPECT_NEAR(h_ret_mat[i][j], h_B_mat[i][j], 1e-3);
        }
    }


    A_mat.release();
    X_mat.release();
    B_mat.release();
    ret_mat.release();
}

TEST(RXMeshStatic, SparseMatrixLowerLevelAPISolve)
{
    using namespace rxmesh;

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");

    uint32_t num_vertices = rx.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto                coords = rx.get_input_vertex_coordinates();
    SparseMatrix<float> A_mat(rx);
    DenseMatrix<float>  X_mat(rx, num_vertices, 3);
    DenseMatrix<float>  B_mat(rx, num_vertices, 3);
    DenseMatrix<float>  ret_mat(rx, num_vertices, 3);

    float time_step = 1.f;

    LaunchBox<threads> launch_box;
    rx.prepare_launch_box(
        {Op::VV}, launch_box, (void*)simple_A_X_B_setup<float, threads>);

    simple_A_X_B_setup<float, threads><<<launch_box.blocks,
                                         launch_box.num_threads,
                                         launch_box.smem_bytes_dyn>>>(
        rx.get_context(), *coords, A_mat, X_mat, B_mat, time_step);

    // A_mat.solve(B_mat, X_mat, Solver::CHOL, PermuteMethod::NSTDIS);
    A_mat.pre_solve(rx, Solver::CHOL, PermuteMethod::NSTDIS);
    A_mat.solve(B_mat, X_mat);

    A_mat.multiply(X_mat, ret_mat);

    std::vector<vec3<float>> h_ret_mat(num_vertices);
    CUDA_ERROR(hipMemcpy(h_ret_mat.data(),
                          ret_mat.data(),
                          num_vertices * 3 * sizeof(float),
                          hipMemcpyDeviceToHost));
    std::vector<vec3<float>> h_B_mat(num_vertices);
    CUDA_ERROR(hipMemcpy(h_B_mat.data(),
                          B_mat.data(),
                          num_vertices * 3 * sizeof(float),
                          hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        for (uint32_t j = 0; j < 3; ++j) {
            EXPECT_NEAR(h_ret_mat[i][j], h_B_mat[i][j], 1e-3);
        }
    }

    A_mat.release();
    X_mat.release();
    B_mat.release();
    ret_mat.release();
}

TEST(RXMeshStatic, SparseMatrixToEigen)
{
    using namespace rxmesh;

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");

    uint32_t num_vertices = rx.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto                coords = rx.get_input_vertex_coordinates();
    SparseMatrix<float> A_mat(rx);
    DenseMatrix<float>  X_mat(rx, num_vertices, 3);
    DenseMatrix<float>  B_mat(rx, num_vertices, 3);


    float time_step = 1.f;

    LaunchBox<threads> launch_box;
    rx.prepare_launch_box(
        {Op::VV}, launch_box, (void*)simple_A_X_B_setup<float, threads>);

    simple_A_X_B_setup<float, threads><<<launch_box.blocks,
                                         launch_box.num_threads,
                                         launch_box.smem_bytes_dyn>>>(
        rx.get_context(), *coords, A_mat, X_mat, B_mat, time_step);

    A_mat.solve(B_mat, X_mat, Solver::CHOL, PermuteMethod::NSTDIS);


    DenseMatrix<float> X_copy(rx, num_vertices, 3);
    X_copy.copy_from(X_mat, DEVICE, HOST);

    A_mat.move(DEVICE, HOST);
    B_mat.move(DEVICE, HOST);

    auto A_eigen = A_mat.to_eigen();
    auto X_eigen = X_mat.to_eigen();
    auto B_eigen = B_mat.to_eigen();

    // Note: there is a bug with Eigen if we use the default reordering
    // which is Eigen::AMDOrdering<int>
    // (https://gitlab.com/libeigen/eigen/-/issues/2839)
    Eigen::SimplicialLDLT<Eigen::SparseMatrix<float>,
                          Eigen::UpLoType::Lower,
                          Eigen::COLAMDOrdering<int>>
        eigen_solver;

    eigen_solver.compute(A_eigen);
    X_eigen = eigen_solver.solve(B_eigen);

    for (int i = 0; i < X_copy.rows(); ++i) {
        for (int j = 0; j < X_copy.cols(); ++j) {
            EXPECT_NEAR(X_eigen(i, j), X_copy(i, j), 0.0000001);
        }
    }

    A_mat.release();
    X_mat.release();
    B_mat.release();
    X_copy.release();
}